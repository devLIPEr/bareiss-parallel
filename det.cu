#include <stdlib.h>
#include <stdio.h>
#include <stdbool.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define epsilon 1e-9
#define MOD 1000000007.0

__device__ static inline int matIndex(int i, int j, int n){
    return i*n+j;
}

__global__ void parallel(double *mat, int n, int n1, int k){
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int stride = blockDim.x*gridDim.x;
    for(int i = k+1+idx; i < n1; i += stride){
        for(int j = k+1; j < n1; j++){
            mat[matIndex(i, j, n1)] = fmod((mat[matIndex(k, k, n1)]*mat[matIndex(i, j, n1)] - mat[matIndex(i, k, n1)]*mat[matIndex(k, j, n1)])/mat[matIndex(k-1, k-1, n1)], MOD);
        }
    }
}

void det(double *mat, float *time, int n, double *cpu_mat, unsigned long long size){
    int sms;
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0);
    printf("SMS: %d\n", sms);
    int n1 = n+1;
    float elapsed;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for(int k = 1; k < n1; k++){
        hipEventRecord(start, 0);
        parallel<<<sms*32, 256>>>(mat, n, n1, k); 
        hipDeviceSynchronize();       
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        *time += elapsed;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    int n, clockRate;
    hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, 0);
    clockRate *= 1e3;
    scanf("%d", &n);
    int n1 = n + 1;

    float time, det_time = 0;
    unsigned long long size = sizeof(double)*n1*n1;
    double *mat = (double*)calloc((n1)*(n1), sizeof(double));
    
    mat[0] = 1;
    for(int i = 1; i < n1; i++){
        for(int j = 1; j < n1; j++){
            scanf("%lf", mat+(i*n1+j));
        }
    }
    
    double *gpu_mat;
    hipMalloc((void **)&gpu_mat, size);
    hipMemcpy(gpu_mat, mat, size, hipMemcpyHostToDevice);

    det(gpu_mat, &det_time, n, mat, size);
    hipMemcpy(mat, gpu_mat, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    printf("Time (DET): %.12lf\n", det_time/clockRate);
    printf("%lf\n", mat[n*n1+n]);
    printf("Time (Total): %.6lf\n", time/1000);
    free(mat);
    hipFree(gpu_mat);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventSynchronize(stop);
    
    return 0;
}